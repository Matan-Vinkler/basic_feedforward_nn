#include "../inc/Matrix.cuh"

Matrix::Matrix(int N, MatrixInitType init_type) : N(N), dev_data(NULL)
{
	std::vector<int> init_data_fill(N * N);

	switch (init_type)
	{
	case MatrixInitType::ZERO:
		std::fill(init_data_fill.begin(), init_data_fill.end(), 0);
		break;
	case MatrixInitType::RANDOM:
		std::generate(init_data_fill.begin(), init_data_fill.end(), []() {return rand() % 100; });
		break;
	default:
		break;
	}

	size_t bytes = N * N * sizeof(int);

	hipMalloc(&dev_data, bytes);
	hipMemcpy(dev_data, init_data_fill.data(), bytes, hipMemcpyHostToDevice);
}

Matrix::Matrix(std::vector<int> data, int N) : N(N), dev_data(NULL)
{
	size_t bytes = N * N * sizeof(int);

	hipMalloc(&dev_data, bytes);
	hipMemcpy(dev_data, data.data(), bytes, hipMemcpyHostToDevice);
}

Matrix::~Matrix()
{
	hipFree(dev_data);
}

void Matrix::matrix_add(Matrix& matrix_a, Matrix& matrix_b)
{
	assert(matrix_a.N == matrix_b.N);
	assert(matrix_a.N == N);

	int THREADS = DEFAULT_THREADS_NUM;
	int BLOCKS = N / THREADS;

	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	matrixAdd << < blocks, threads >> > (matrix_a.dev_data, matrix_b.dev_data, dev_data, N);
}

void Matrix::matrix_scale(Matrix& matrix_a, int factor_scale)
{
	assert(matrix_a.N == N);

	int THREADS = DEFAULT_THREADS_NUM;
	int BLOCKS = N / THREADS;

	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	matrixScale << < blocks, threads >> > (matrix_a.dev_data, factor_scale, dev_data, N);
}

void Matrix::matrix_mul(Matrix& matrix_a, Matrix& matrix_b)
{
	assert(matrix_a.N == matrix_b.N);
	assert(matrix_a.N == N);

	int THREADS = DEFAULT_THREADS_NUM;
	int BLOCKS = N / THREADS;

	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	matrixMul << < blocks, threads >> > (matrix_a.dev_data, matrix_b.dev_data, dev_data, N);
}

std::vector<int> Matrix::export_to_host()
{
	std::vector<int> h_data(N * N);
	size_t bytes = N * N * sizeof(int);

	hipMemcpy(h_data.data(), dev_data, bytes, hipMemcpyDeviceToHost);

	return h_data;
}
