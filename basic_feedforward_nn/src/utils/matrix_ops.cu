#include "hip/hip_runtime.h"
#include "../../inc/utils/matrix_ops.cuh"

__global__ void matrixMul(const int* a, const int* b, int* c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++)
    {
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

__global__ void matrixAdd(const int* a, const int* b, int* c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = a[row * N + col] + b[row * N + col];
}

__global__ void matrixScale(const int* a, int factor, int* c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = a[row * N + col] * factor;
}